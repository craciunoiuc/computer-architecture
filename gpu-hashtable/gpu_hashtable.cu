#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>

#include "./gpu_hashtable.hpp"

/*
 * The operations in each for iteration are atomic,
 * so there won't be any concurrency problems.
 * After the Compare-And-Set operation there will be 3 cases:
 * 1. The returned value (old value) is 0 -> the value will be added
 * 2. The returned value is the same as the key -> the value will be replaced
 * 3. The returned value is not the same as the key -> the entry is skipped
 */
__global__ void insert_entry(int *keys, int *values, int nr_keys,
							hashtable_t hashtable) {
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	CHECK_ERROR_NORET(idx >= nr_keys);
	CHECK_ERROR_NORET(keys[idx] <= 0 || values[idx] <= 0);
	
	// Save the key that needs to be added and calculate its hash
	int key_to_add = keys[idx];
	int hash = HASH_INT(key_to_add, hashtable.max_elements);

	// Iterate through the table until an empty spot is
	// found in [hash, max_elements)
	for (int i = hash; i < hashtable.max_elements; ++i) {
		int key_before = atomicCAS(&hashtable.entries[i].key,
								KEY_INVALID, key_to_add);
		if (key_before == KEY_INVALID || key_before == key_to_add) {
			atomicExch(&hashtable.entries[i].value, values[idx]);
			return;
		}
	}

	// Iterate through the table in a similar fashion,
	// until an empty spot is found in [0, hash)
	for (int i = 0; i < hash; ++i) {
		int key_before = atomicCAS(&hashtable.entries[i].key,
									KEY_INVALID, key_to_add);
		if (key_before == KEY_INVALID || key_before == key_to_add) {
			atomicExch(&hashtable.entries[i].value, values[idx]);
			return;
		}
	}

	// The end should not be reached (the function gets here if table is full)
}

/*
 * The function (for each thread) iterates through the table and writes the
 * first value that matches the key into the values vector
 */
__global__ void get_entry(int *keys, int *values, int nr_keys,
							hashtable_t hashtable) {
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	CHECK_ERROR_NORET(idx >= nr_keys);
	int key_to_find = keys[idx];
	int hash = HASH_INT(key_to_find, hashtable.max_elements);

	for (int i = hash; i < hashtable.max_elements; ++i) {
		if (hashtable.entries[i].key == key_to_find) {
			atomicExch(&values[idx], hashtable.entries[i].value);
			return;
		}
	}
	
	for (int i = 0; i < hash; ++i) {
		if (hashtable.entries[i].key == key_to_find) {
			atomicExch(&values[idx], hashtable.entries[i].value);
			return;
		}
	}

	// Should get here only if the value is not found
	values[idx] = -1;
}

/*
 * The function works the same as the insert function. The function is called
 * for each element of the old hashtable and each thread inserts its element
 * in the hashtable in a similar fashion as insert_entry.
 */
__global__ void reshape_entry(entry_t *old_entries, int old_max_elements,
							  entry_t *new_entries, int new_max_elements) {
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;				  
	CHECK_ERROR_NORET(idx >= old_max_elements);
	CHECK_ERROR_NORET(old_entries[idx].key == KEY_INVALID);
	
	int key_to_add = old_entries[idx].key;
	int hash = HASH_INT(key_to_add, new_max_elements);

	// Iterate through the table until an empty spot is
	// found in [hash, max_elements)
	for (int i = hash; i < new_max_elements; ++i) {
		int key_before = atomicCAS(&new_entries[i].key,
								KEY_INVALID, key_to_add);
		if (key_before == KEY_INVALID) {
			atomicExch(&new_entries[i].value, old_entries[idx].value);
			return;
		}
	}

	// Iterate through the table in a similar fashion,
	// until an empty spot is found in [0, hash)
	for (int i = 0; i < hash; ++i) {
		int key_before = atomicCAS(&new_entries[i].key,
									KEY_INVALID, key_to_add);
		if (key_before == KEY_INVALID) {
			atomicExch(&new_entries[i].value, old_entries[idx].value);
			return;
		}
	}

	// End should be reached if the table is full, impossible in this case
}

/* 
 * The hashtable constructor. Initialises the hashtable dimensions and clears
 * the VRAM memory.
 */
GpuHashTable::GpuHashTable(int size) {
	hipError_t error_code;

	hashtable.num_elements = 0;
	hashtable.max_elements = size;
	hashtable.entries = nullptr;

	error_code = hipMalloc(&hashtable.entries, size * sizeof(entry_t));
	DIE(error_code != hipSuccess, "Failed to Allocate VRAM");
	error_code = hipMemset(hashtable.entries, 0, size * sizeof(entry_t));
	DIE(error_code != hipSuccess, "Failed to clear Allocated VRAM");
}

/*
 * Frees the VRAM memory.
 */
GpuHashTable::~GpuHashTable() {
	DIE(hipFree(hashtable.entries) != hipSuccess, "Failed to free hashtable");
}

/*
 * Function that allocates a new hashtable and inserts all the elements from
 * the old one into the new one, rehashing them.
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	entry_t *new_entries;
	int new_max_elements = numBucketsReshape;
	hipError_t error_code;

	// Check for valid input
	CHECK_ERROR_NORET(!numBucketsReshape);

	// Allocate new memory for the hashtable and clear it
	error_code = hipMalloc(&new_entries, new_max_elements * sizeof(entry_t));
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));
	error_code = hipMemset(new_entries, 0, new_max_elements * sizeof(entry_t));
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));

	// Calculate the number of blocks and call de GPU function
	int num_blocks = hashtable.max_elements % BLOCK_THREADS ?
					 hashtable.max_elements / BLOCK_THREADS + 1 :
					 hashtable.max_elements / BLOCK_THREADS;
	reshape_entry<<<num_blocks, BLOCK_THREADS>>>(hashtable.entries,
					hashtable.max_elements, new_entries, new_max_elements);

	// Ensure that all threads finished
	error_code = hipDeviceSynchronize();
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));

	// Free the old memory
	error_code = hipFree(hashtable.entries);
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));
	
	// Save the new hashtable
	hashtable.entries = new_entries;
	hashtable.max_elements = new_max_elements;
}

/*
 * Function to insert a numKeys batch of key-value pairs in the GPU hashtable.
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	int *gpu_keys, *gpu_values;
	hipError_t error_code;

	// Check for invalid input
	CHECK_ERROR(!keys || !values || !numKeys, false);

	// Resize the hashtable to 0.8f if the new load factor is above 0.85f
	if (static_cast<float>(hashtable.num_elements + numKeys) /
		static_cast<float>(hashtable.max_elements) > HARD_LIMIT) {
			
		reshape(static_cast<float>(hashtable.num_elements +
				numKeys) / SOFT_LIMIT);
	}

	// Allocate video memory
	error_code = hipMalloc(&gpu_keys, numKeys * sizeof(int));
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));
	error_code = hipMalloc(&gpu_values, numKeys * sizeof(int));
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));

	// Copy information into the video memory
	error_code = hipMemcpy(gpu_keys, keys, numKeys * sizeof(int),
					hipMemcpyHostToDevice);
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));
	error_code = hipMemcpy(gpu_values, values, numKeys * sizeof(int),
					hipMemcpyHostToDevice);
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));

	// Calculate number of GPU blocks needed and call the GPU function
	int num_blocks = numKeys % BLOCK_THREADS ?
					 numKeys / BLOCK_THREADS + 1 :
					 numKeys / BLOCK_THREADS;
	insert_entry<<<num_blocks, BLOCK_THREADS>>>(gpu_keys, gpu_values,
												numKeys, hashtable);

	// Ensure that all CUDA threads finish before freeing the memory
	error_code = hipDeviceSynchronize();
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));
	hashtable.num_elements += numKeys;

	// Free memory
	error_code = hipFree(gpu_keys);
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));
	error_code = hipFree(gpu_values);
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));

	return true;
}

/*
 * Function to retrieve numKeys elements from the GPU hashtable.
 */
int *GpuHashTable::getBatch(int* keys, int numKeys) {
	int *gpu_keys, *gpu_values, *result;
	hipError_t error_code;

	// Check parameters validity
	CHECK_ERROR(!keys || !numKeys, nullptr);

	// Allocate memory in RAM & VRAM
	result = (int *)malloc(numKeys * sizeof(int));
	DIE(!result, "Malloc");
	error_code = hipMalloc(&gpu_keys, numKeys * sizeof(int));
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));
	error_code = hipMalloc(&gpu_values, numKeys * sizeof(int));
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));

	// Clear allocated memory and initialize it
	error_code = hipMemset(gpu_values, -1, numKeys * sizeof(int));
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));
	error_code = hipMemcpy(gpu_keys, keys, numKeys * sizeof(int),
							hipMemcpyHostToDevice);
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));

	// Calculate number of GPU blocks needed and call the GPU function
	int num_blocks = numKeys % BLOCK_THREADS ?
					 numKeys / BLOCK_THREADS + 1 :
					 numKeys / BLOCK_THREADS;
	get_entry<<<num_blocks, BLOCK_THREADS>>>(gpu_keys, gpu_values,
											 numKeys, hashtable);

	// Make sure all devices finished finding keys
	error_code = hipDeviceSynchronize();
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));

	// Copy the results back in memory
	error_code = hipMemcpy(result, gpu_values, numKeys * sizeof(int),
							hipMemcpyDeviceToHost);
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));

	// Free the VRAM memory
	error_code = hipFree(gpu_keys);
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));
	error_code = hipFree(gpu_values);
	DIE(error_code != hipSuccess, hipGetErrorString(error_code));

	return result;
}

/*
 * Computes the floating point division between the current number of elements
 * and the maximum number of elements in the hashtable.
 */
float GpuHashTable::loadFactor() {
	return static_cast<float>(hashtable.num_elements) /
		static_cast<float>(hashtable.max_elements);
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "./test_map.cpp"
